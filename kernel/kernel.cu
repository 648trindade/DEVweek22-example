#include "hip/hip_runtime.h"
#include "kernel.h"

#include <cstdint>

__host__ __device__ void move(float3 &position) {
    constexpr float3 velocity {0, -1, 0};
    constexpr float dt {1e-5f};

    position.x = velocity.x * dt;
    position.y = velocity.y * dt;
    position.z = velocity.z * dt;
}

__global__ void dispatch_move_kernel(float3 *positions, uint32_t size) {
    const uint32_t start = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t step = blockDim.x;

    for (uint32_t i = start; i < size; i += step) {
        move(positions[i]);
    }
}

void cuda_move(std::vector<float3> &positions) {
    float3 *device_positions;
    hipMalloc(&device_positions, positions.size() * sizeof(float3));
    hipMemcpy(device_positions, positions.data(),
               positions.size() * sizeof(float3), hipMemcpyHostToDevice);

    dispatch_move_kernel<<<1, 1>>>(device_positions, positions.size());

    hipMemcpy(positions.data(), device_positions,
               positions.size() * sizeof(float3), hipMemcpyDeviceToHost);
    hipFree(device_positions);
}

void omp_move(std::vector<float3> &positions) {
    #pragma omp parallel for
    for (int i = 0; i < positions.size(); i++) {
        move(positions[i]);
    }
}

void move_positions(std::vector<float3> &positions, bool use_cuda) {
    if (use_cuda) {
        cuda_move(positions);
    } else {
        omp_move(positions);
    }
}
