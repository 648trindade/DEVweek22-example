#include "hip/hip_runtime.h"
#include "kernel/kernel.h"

#include <cmath>
#include <iostream>
#include <omp.h>

int main(int argc, char *argv[]) {
    const unsigned int n_positions = std::atoi(argv[1]);
    const bool use_cuda = (argc == 3) && bool(std::atoi(argv[2]));

    srand(0);

    std::vector<float3> positions(n_positions);
    for (unsigned int i = 0; i < n_positions; i++) {
        positions[i] = {
                1000.f * float(rand()) / RAND_MAX,
                1000.f * float(rand()) / RAND_MAX,
                1000.f * float(rand()) / RAND_MAX
        };
    }

    double start = omp_get_wtime();
    move_positions(positions, use_cuda);
    double end = omp_get_wtime();

    std::cout << "Duration: " << (end - start) << " s" << std::endl;
    return 0;
}
